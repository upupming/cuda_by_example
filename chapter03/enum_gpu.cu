/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */

#include "../common/book.h"

int main(void) {
    hipDeviceProp_t prop;

    int count;
    HANDLE_ERROR(hipGetDeviceCount(&count));
    for (int i = 0; i < count; i++) {
        HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
        printf("   --- General Information for device %d ---\n", i);
        printf("Name:  %s\n", prop.name);
        printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
        printf("Clock rate:  %d\n", prop.clockRate);
        printf("Device copy overlap:  ");
        if (prop.deviceOverlap)
            printf("Enabled\n");
        else
            printf("Disabled\n");
        printf("Kernel execution timeout :  ");
        if (prop.kernelExecTimeoutEnabled)
            printf("Enabled\n");
        else
            printf("Disabled\n");

        printf("   --- Memory Information for device %d ---\n", i);
        printf("Total global mem:  %lld\n", prop.totalGlobalMem);
        printf("Total constant Mem:  %lld\n", prop.totalConstMem);
        printf("Max mem pitch:  %lld\n", prop.memPitch);
        printf("Texture Alignment:  %ld\n", prop.textureAlignment);

        printf("   --- MP Information for device %d ---\n", i);
        printf("Multiprocessor count:  %d\n",
               prop.multiProcessorCount);
        printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
        printf("Registers per mp:  %d\n", prop.regsPerBlock);
        printf("Threads in warp:  %d\n", prop.warpSize);
        printf("Max threads per block:  %d\n",
               prop.maxThreadsPerBlock);
        printf("Max thread dimensions:  (%d, %d, %d)\n",
               prop.maxThreadsDim[0], prop.maxThreadsDim[1],
               prop.maxThreadsDim[2]);
        printf("Max grid dimensions:  (%lld, %d, %d)\n",
               prop.maxGridSize[0], prop.maxGridSize[1],
               prop.maxGridSize[2]);
        printf("\n");
    }
}
/*
   --- General Information for device 0 ---
Name:  GeForce GTX 980 Ti
Compute capability:  5.2
Clock rate:  1354500
Device copy overlap:  Enabled
Kernel execution timeout :  Enabled
   --- Memory Information for device 0 ---
Total global mem:  6442450944
Total constant Mem:  65536
Max mem pitch:  2147483647
Texture Alignment:  512
   --- MP Information for device 0 ---
Multiprocessor count:  22
Shared mem per mp:  49152
Registers per mp:  65536
Threads in warp:  32
Max threads per block:  1024
Max thread dimensions:  (1024, 1024, 64)
Max grid dimensions:  (2147483647, 65535, 65535)
*/
